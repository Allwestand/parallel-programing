#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#define N 8

//Interleave addressing kernael_version
__global__ void interleaved_reduce(int* d_in,int* d_out)
{
     int i= threadIdx.x;
    // int M= N/2;
    // for(int s=1; s<N; s=s<<1){
//            if(i<M) {
//              printf("stride = %d, thread %d, is active \n",s,i);
//              d_in[(2*s)*i]=d_in[(2*s)*i] + d_in[(2*s)*i+s];

//            }
 //           M= M/2;

   //  }
    // if(i == 0)
     //         d_out[0] = d_in[0];

         __shared__  int temp[N];
         int idx = threadIdx.x + blockIdx.x*blockDim.x;
         temp[i] = d_in[idx];
         for(int s=1; s<blockDim.x ; s=s<<1){
               if(i<blockDim.x){
                       printf("stride = %d, thread %d, is active \n",s,i);
                       temp[(2*s)*i] = temp[(s*2)*i] + temp[(2*s)*i+s];

               }
                __syncthreads();





         }
         if(i ==0)
                 d_out[blockIdx.x] = temp[0];
}

//Contiguous addressing kernel version
__global__ void contiguous_reduce(int* d_in,int* d_out)
{



           //  int i= threadIdx.x;
           //  int M= N/2;
           //  for(int s=M; s>0; s=s>>1){
        //       if(i<M) {
        //              printf("stride = %d, thread %d, is active \n",s,i);
        //               d_in[i]=d_in[i] + d_in[i+s];

        //               }
        //       M= M/2;
 // }
//           if(i == 0)
//               d_out[0] = d_in[0];

    //share memory
        __shared__ int temp[N];
	 int i  = threadIdx.x;
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        temp[i]=d_in[idx];
       __syncthreads();
        for(int s=blockDim.x; s>0;s=s>>1){
                if(i<s){
                        printf("stride = %d, thread %d, is active \n",s,i);
                        temp[i]=temp[i+s];

        }



        if(i==0)
            d_out[blockIdx.x] = temp[0];


        }
}

int main()
{

        int h_in[N];
        int h_out= 0;

        for(int i=0; i<N; i++){
                h_in[i] = i+1;

        int *d_in, *d_out;
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipMalloc((void**) &d_in, N*sizeof(int));
        hipMalloc((void**) &d_out, sizeof(int));
        hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);
        //kernel call
        hipEventRecord(start);
        //interleaved_reduce<<<1,1024>>>(d_in,d_out);
        contiguous_reduce<<<1,1024>>>(d_in, d_out);

        hipEventRecord(stop);
        hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_in);
        hipFree(d_out);

        hipEventSynchronize(stop);
        float Timeused;
        hipEventElapsedTime(&Timeused, start , stop);
        printf("Output: %d \n Time used: %f ", h_out, Timeused);




        return -1;

        }

