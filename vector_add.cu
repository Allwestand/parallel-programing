
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void parallel_vector_add(int* d_a, int* d_b, int* d_c,int *d_n )
{

	int i = (blockIdx.x*blockIdx.x)+threadIdx.x;
	if(i < *d_n){
		printf(" I am thread #%d, and about to compute c[%d]. \n",i,i);
		d_c[i]=d_a[i]+d_b[i];
	}
	else{
		printf("I am thread #%d, and doing nothing.\n" , i);
	}
}


int main()
{
	int n;
	scanf("%d" , &n);

        // declare input and output on host
	int h_a[n];
	int h_b[n];
	int h_c[n];
	for(int i=0;i<n;i++)
	{
	    h_a[i] =i;
	    h_b[i] = n-i;
	
	}

	int* d_a, *d_b, *d_c, *d_n;
	hipMalloc((void **) &d_a, n*sizeof(int));
	hipMalloc((void **) &d_b, n*sizeof(int));
	hipMalloc((void **) &d_c, n*sizeof(int));
	hipMalloc((void **) &d_n, sizeof(int));

	hipMemcpy(d_a, &h_a, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

	// Part II: Kernel Launch
	int max=(n/512);
	parallel_vector_add<<<max,512>>>(d_a,d_b,d_c,d_n);
        hipDeviceSynchronize();

	//Part III: Copy data from device back to host. and free all data allocate on devicce
	hipMemcpy(&h_c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
       
	for(int i=0;i<n;i++)
	    printf("%d ", h_c[i]);




}
