
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define N 8

__global__ void exclusive_scan(int *d_in)
{
            __shared__ int  temp_in[N];
            int i = threadIdx.x;
            temp_in[i]=d_in[i];
            //Phase 1 (Uptree)
            int s = 1;
            for(; s<=N-1; s<<=1)
         {
                     int i = 2*s*(threadIdx.x+1)-1;
                     if((i-s >= 0) && (i<N)) {
                        //printf("s = %d, i= %d \n", s, i);
                         int a = temp_in[i];
                         int b = temp_in[i-s];
                         __syncthreads();
                         temp_in[i] = a+b;

                         //printf("Write in[%d] = %d\n", i, a+b);
         }
                  __syncthreads();

          }

                    //Phase 2 (Downtree)
                    if(threadIdx.x == 0)
                          temp_in[N-1] = 0;

                    for(s = s/2; s >= 1; s>>=1)
                    {
                       int i = 2*s*(threadIdx.x+1)-1;
                       if((i-s >= 0) && (i<N)) {
                              //printf("s = %d, i= %d \n", s, i);
                              int r = temp_in[i];
                              int l = temp_in[i-s];
                               __syncthreads();
                              temp_in[i] = l+r;
                              temp_in[i-s] = r;
                              __syncthreads();
                              //printf("Write in[%d] = %d\n", i, a+b);
                                                                     }
                         __syncthreads();
                    }
                    d_in[i]=temp_in[i];
}

int main()
{
                int h_in[N];
                int h_out[N];

                h_in[0] = 3;
                h_in[1] = 1;
                h_in[2] = 7;
                h_in[3] = 0;
                h_in[4] = 4;
                h_in[5] = 1;
                h_in[6] = 6;
                h_in[7] = 3;

                int *d_in;
               //int *d_out;
                hipEvent_t start,stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);

              hipMalloc((void**) &d_in, N*sizeof(int));
             //cudaMalloc((void**) &d_out, N*sizeof(int));
             hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);
             hipEventRecord(start);
             //Implementing kernel call
            exclusive_scan<<<1, 4>>>(d_in);
            hipEventRecord(stop);
            hipMemcpy(&h_out, d_in, N*sizeof(int), hipMemcpyDeviceToHost);
            hipFree(d_in);
            hipEventSynchronize(stop);
            float TimeUsed;
            hipEventElapsedTime(&TimeUsed, start ,stop);
            for(int i=0; i<N; i++)
                 printf("out[%d] =  %d\n", i, h_out[i]);
            printf("Time used: %f\n ",TimeUsed);
            //cudaFree(d_in);
            return -1;

}

