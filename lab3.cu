
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 30
void add(int *X,int *Y,int *Z)
{
        for(int i=0;i<N;i++)
            for(int j=0;j<N;j++)
                    Z[i*N+j] = X[i*N+j]+Y[i*N+j];

}

__global__ void add_kernel(int *X,int *Y,int *Z)
{
        int i = threadIdx.x;
        int j = threadIdx.y;

        Z[i*N+j] = X[i*N+j]+Y[i*N+j];

}

int main()
{
        int X[N*N];
        int Y[N*N];

        for(int i=0;i<N;i++)
           for(int j=0;j<N;j++) {
                X[i*N+j]=-1;
                Y[i*N+j]= 1;
           }

        //Outout matrix
        int Z[N*N];

        int *d_X,*d_Y,*d_Z;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipMalloc((void**) &d_X, (N*N)*sizeof(int));
        hipMalloc((void**) &d_Y, (N*N)*sizeof(int));
        hipMalloc((void**) &d_Z, (N*N)*sizeof(int));

        hipMemcpy(d_X, &X,(N*N)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Y, &Y,(N*N)*sizeof(int), hipMemcpyHostToDevice);


        dim3 dimGrid(13,13,1);
        dim3 dimBlock(22,22,1);
        hipEventRecord(start);
        add_kernel<<<dimGrid, dimBlock>>>(d_X,d_Y,d_Z);
        hipEventRecord(stop);
        //add(X, Y, Z);

        hipMemcpy(&Z, d_Z,(N*N)*sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_X);
        hipFree(d_Y);
        hipFree(d_Z);
        hipEventSynchronize(stop);
        float Timeused = 0;
        hipEventElapsedTime(&Timeused, start, stop);
        for(int i=0;i<N;i++)
                for(int j=0;j<N;j++){
                printf("%d ",Z[i*N+j]);
	  }
                printf("\n");
                printf("Time used:%f ",Timeused);
                return -1;

}

